#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Periodic shuffle operation
// Copyright (c) 2016 Georgia Tech
// Licensed under The MIT License 
// Written by Yi Li
// ------------------------------------------------------------------

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/periodic_shuffle_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PeriodicShuffleFwdKernel(const int nthreads,
  const Dtype* bottom_data, 
  const int width, const int height, const int channels, const int group_size, 
  const int shuffled_width, const int shuffled_height, const int shuffled_channels,
  Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // n1=n, c1, h1, w1
    int n = index / width / height / channels;
    int c1 = (index / width / height) % channels;
    int h1 = (index / width) % height;
    int w1 = index % width;
    // -> n2=n, c2, h2, w2
    int c2 = c1 % shuffled_channels;
    int h2 = group_size * h1 + (c1 / shuffled_channels) / group_size;
    int w2 = group_size * w1 + (c1 / shuffled_channels) % group_size;
      // -> idx
    int idx = n * channels * width * height + 
              c2 * shuffled_width * shuffled_height + h2 * shuffled_width + w2;
    top_data[idx] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void PeriodicShuffleBwdKernel(const int nthreads,
  const Dtype* top_diff, 
  const int width, const int height, const int channels, const int group_size, 
  const int shuffled_width, const int shuffled_height, const int shuffled_channels,
  Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // n1=n, c1, h1, w1
    int n = index / width / height / channels;
    int c1 = (index / width / height) % channels;
    int h1 = (index / width) % height;
    int w1 = index % width;
    // -> n2=n, c2, h2, w2
    int c2 = c1 % shuffled_channels;
    int h2 = group_size * h1 + (c1 / shuffled_channels) / group_size;
    int w2 = group_size * w1 + (c1 / shuffled_channels) % group_size;
      // -> idx
    int idx = n * channels * width * height + 
              c2 * shuffled_width * shuffled_height + h2 * shuffled_width + w2;
    bottom_diff[index] = top_diff[idx];
  }
}
  
template <typename Dtype>
void PeriodicShuffleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  PeriodicShuffleFwdKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, width_, height_, channels_, group_size_,
      shuffled_width_, shuffled_height_, shuffled_channels_, top_data);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void PeriodicShuffleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  PeriodicShuffleBwdKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, width_, height_, channels_, group_size_,
      shuffled_width_, shuffled_height_, shuffled_channels_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(PeriodicShuffleLayer);

}  // namespace caffe